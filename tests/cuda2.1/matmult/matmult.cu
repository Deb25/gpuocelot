/*!
	\file matmult.cu
	
	\author Andrew Kerr <arkerr@gatech.edu>

	\brief use whatever you have to achieve fast (~150 GFLOP/s) matrix multiply on a GPU
*/



#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>
#define CEIL_DIV(a,b) (((a) == ((a)/(b)) * (b)) ?  ((a)/(b)):((a)/(b)+1) )


///////////////////////////////////////////////////////////////////////////////////////////////////

/*!
	Performs matrix multiplication
*/
__global__ void k_matMult_naive(
	float *A,
	float *B,
	float *C,
	int A_rows,
	int A_cols,
	int B_cols) {

#define BW 16

	int k;

	__shared__ float A_block[BW][BW];
	__shared__ float B_block[BW][BW];


	float *a_ptr = A + (blockIdx.y * blockDim.y + threadIdx.y) * A_cols + threadIdx.x;
	float *b_ptr = B + blockIdx.x * blockDim.x + threadIdx.x + threadIdx.y * B_cols;

	float c_v = 0;

	for (k = 0; k < A_cols; k += BW) {
		int j;

		A_block[threadIdx.y][threadIdx.x] = *a_ptr;
		__syncthreads();
		B_block[threadIdx.y][threadIdx.x] = *b_ptr;
		__syncthreads();

		for (j = 0; j < BW; j++) {
			c_v += A_block[threadIdx.y][j] * B_block[j][threadIdx.x];
		}
		__syncthreads();
		a_ptr += BW;
		b_ptr += BW * B_cols;
	}

	
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	C[row * B_cols + col] = c_v;
}

void matrix_multiply_naive(float *A, float *B, float *C, int A_rows, int A_cols, int B_cols) {
	dim3 grid(A_rows / 16, B_cols / 16);
	dim3 block(16, 16);
	k_matMult_naive<<< grid, block >>>(A, B, C, A_rows, A_cols, B_cols);
}

///////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void k_matMult_unroll(
  float *A,
  float *B,
  float *C,
  int A_rows,
  int A_cols,
  int B_cols) {

#define BW 16

  int k;

  __shared__ float A_block[BW][BW];
  __shared__ float B_block[BW][BW*2];


  float *a_ptr = A + (blockIdx.y * blockDim.y + threadIdx.y) * A_cols + threadIdx.x;
  float *b_ptr = B + 2 * blockIdx.x * blockDim.x + threadIdx.x + threadIdx.y * B_cols;

  float c_v0 = 0;
	float c_v1 = 0;

  for (k = 0; k < A_cols; k += BW) {

    A_block[threadIdx.y][threadIdx.x] = *a_ptr;
    __syncthreads();
    B_block[threadIdx.y][threadIdx.x] = b_ptr[0];
		B_block[threadIdx.y][threadIdx.x+blockDim.x] = b_ptr[blockDim.x];
    __syncthreads();

		int X = threadIdx.x;
		int Y = threadIdx.y;

    float a00 = A_block[Y][0];
    float a01 = A_block[Y][1];
    float a02 = A_block[Y][2];
    float a03 = A_block[Y][3];
    float a04 = A_block[Y][4];
    float a05 = A_block[Y][5];
    float a06 = A_block[Y][6];
    float a07 = A_block[Y][7];
    float a08 = A_block[Y][8];
    float a09 = A_block[Y][9];
    float a10 = A_block[Y][10];
    float a11 = A_block[Y][11];
    float a12 = A_block[Y][12];
    float a13 = A_block[Y][13];
    float a14 = A_block[Y][14];
    float a15 = A_block[Y][15];

		c_v0 += a00 * B_block[0][X];
		c_v0 += a01 * B_block[1][X];
		c_v0 += a02 * B_block[2][X];
		c_v0 += a03 * B_block[3][X];
		c_v0 += a04 * B_block[4][X];
		c_v0 += a05 * B_block[5][X];
		c_v0 += a06 * B_block[6][X];
		c_v0 += a07 * B_block[7][X];
		c_v0 += a08 * B_block[8][X];
		c_v0 += a09 * B_block[9][X];
		c_v0 += a10 * B_block[10][X];
		c_v0 += a11 * B_block[11][X];
		c_v0 += a12 * B_block[12][X];
		c_v0 += a13 * B_block[13][X];
		c_v0 += a14 * B_block[14][X];
		c_v0 += a15 * B_block[15][X];

		X = threadIdx.x + BW;

    c_v1 += a00 * B_block[0][X];
		c_v1 += a01 * B_block[1][X];
    c_v1 += a02 * B_block[2][X];
		c_v1 += a03 * B_block[3][X];
    c_v1 += a04 * B_block[4][X];
		c_v1 += a05 * B_block[5][X];
    c_v1 += a06 * B_block[6][X];
		c_v1 += a07 * B_block[7][X];
    c_v1 += a08 * B_block[8][X];
		c_v1 += a09 * B_block[9][X];
    c_v1 += a10 * B_block[10][X];
		c_v1 += a11 * B_block[11][X];
    c_v1 += a12 * B_block[12][X];
		c_v1 += a13 * B_block[13][X];
    c_v1 += a14 * B_block[14][X];
		c_v1 += a15 * B_block[15][X];

    __syncthreads();
    a_ptr += BW;
    b_ptr += BW * B_cols;
  }

  int col = blockIdx.x * 2 * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  C[row * B_cols + col] = c_v0;
	C[row * B_cols + col + blockDim.x] = c_v1;
}

void matrix_multiply_unroll(float *A, float *B, float *C, int A_rows, int A_cols, int B_cols) {
  dim3 grid(B_cols / 32, A_rows / 16);
  dim3 block(16, 16);
  k_matMult_unroll<<< grid, block >>>(A, B, C, A_rows, A_cols, B_cols);
}

///////////////////////////////////////////////////////////////////////////////////////////////////
//
// Macros used by k_matMul_unroll2

#define INNER_PROD(cc) { \
		cc += a00 * B_block[0][X];	\
		cc += a01 * B_block[1][X];	\
		cc += a02 * B_block[2][X];	\
		cc += a03 * B_block[3][X];	\
		cc += a04 * B_block[4][X];	\
		cc += a05 * B_block[5][X];	\
		cc += a06 * B_block[6][X];	\
		cc += a07 * B_block[7][X];	\
		cc += a08 * B_block[8][X];	\
		cc += a09 * B_block[9][X];	\
		cc += a10 * B_block[10][X];	\
		cc += a11 * B_block[11][X];	\
		cc += a12 * B_block[12][X];	\
		cc += a13 * B_block[13][X];	\
		cc += a14 * B_block[14][X];	\
		cc += a15 * B_block[15][X];	\
	}

#define LOAD_A_REGS() {		\
		a00 = A_block[Y][0];	\
    a01 = A_block[Y][1];	\
    a02 = A_block[Y][2];	\
    a03 = A_block[Y][3];	\
    a04 = A_block[Y][4];	\
    a05 = A_block[Y][5];	\
    a06 = A_block[Y][6];	\
    a07 = A_block[Y][7];	\
    a08 = A_block[Y][8];	\
    a09 = A_block[Y][9];	\
    a10 = A_block[Y][10];	\
    a11 = A_block[Y][11];	\
    a12 = A_block[Y][12];	\
    a13 = A_block[Y][13];	\
    a14 = A_block[Y][14];	\
    a15 = A_block[Y][15];	\
}

///////////////////////////////////////////////////////////////////////////////////////////////////
/*
	Assumes A, B, and C are unit-stride and row-major. A_rows, A_cols, B_cols are all divisible by 64
*/
__global__ void k_matMult_unroll2(
  float *A,
  float *B,
  float *C,
  int A_rows,
  int A_cols,
  int B_cols) {

#define BW 16

  int k;

  __shared__ float A_block[BW*4][BW];
  __shared__ float B_block[BW][BW*4];

	int col = blockIdx.x * blockDim.x * 4 + threadIdx.x;
  int row = 4 * blockIdx.y * blockDim.y + threadIdx.y;
	C += row * B_cols + col;

  float *a_ptr = A + row * A_cols + threadIdx.x;
  float *b_ptr = B + col + threadIdx.y * B_cols;

  float c_v00 = 0;
	float c_v01 = 0;
	float c_v02 = 0;
	float c_v03 = 0;
  float c_v10 = 0;
	float c_v11 = 0;
	float c_v12 = 0;
	float c_v13 = 0;
	
  float c_v20 = 0;
	float c_v21 = 0;
	float c_v22 = 0;
	float c_v23 = 0;
  float c_v30 = 0;
	float c_v31 = 0;
	float c_v32 = 0;
	float c_v33 = 0;


  for (k = 0; k < A_cols; k += BW) {

    A_block[threadIdx.y][threadIdx.x] = a_ptr[0];
		A_block[threadIdx.y + BW][threadIdx.x] = a_ptr[BW*A_cols];
    __syncthreads();
    B_block[threadIdx.y][threadIdx.x] = b_ptr[0];
		B_block[threadIdx.y][threadIdx.x+blockDim.x] = b_ptr[blockDim.x];
		B_block[threadIdx.y][threadIdx.x+blockDim.x*2] = b_ptr[blockDim.x*2];
		B_block[threadIdx.y][threadIdx.x+blockDim.x*3] = b_ptr[blockDim.x*3];
		A_block[threadIdx.y + BW*2][threadIdx.x] = a_ptr[BW*A_cols*2];
		A_block[threadIdx.y + BW*3][threadIdx.x] = a_ptr[BW*A_cols*3];
		__syncthreads();

		int X, Y;
		float a00, a01, a02, a03, a04, a05, a06, a07;
		float a08, a09, a10, a11, a12, a13, a14, a15;

		Y = threadIdx.y;
		LOAD_A_REGS();
		X = threadIdx.x;
		INNER_PROD(c_v00);

		X += BW;
		INNER_PROD(c_v01);
		
		X += BW;
		INNER_PROD(c_v02);

		X += BW;
		INNER_PROD(c_v03);
		
		__syncthreads();
		
		Y += BW;
		LOAD_A_REGS();

		X = threadIdx.x;
		INNER_PROD(c_v10);

		X += BW;
		INNER_PROD(c_v11);
		
		X += BW;
		INNER_PROD(c_v12);

		X += BW;
		INNER_PROD(c_v13);

		__syncthreads();
		
		Y += BW;
		LOAD_A_REGS();
		
		X = threadIdx.x;
		INNER_PROD(c_v20);

		X += BW;
		INNER_PROD(c_v21);
		
		X += BW;
		INNER_PROD(c_v22);

		X += BW;
		INNER_PROD(c_v23);

		__syncthreads();
		
		Y += BW;
		LOAD_A_REGS();
		
		X = threadIdx.x;
		INNER_PROD(c_v30);

		X += BW;
		INNER_PROD(c_v31);
		
		X += BW;
		INNER_PROD(c_v32);

		X += BW;
		INNER_PROD(c_v33);

    a_ptr += BW;
    b_ptr += BW * B_cols;
  }

  C[0] = c_v00;
	C[blockDim.x] = c_v01;
  C[blockDim.x*2] = c_v02;
	C[blockDim.x*3] = c_v03;
	
	C += B_cols * BW;
  C[0] = c_v10;
	C[blockDim.x] = c_v11;
  C[blockDim.x*2] = c_v12;
	C[blockDim.x*3] = c_v13;
	
	C += B_cols * BW;
  C[0] = c_v20;
	C[blockDim.x] = c_v21;
  C[blockDim.x*2] = c_v22;
	C[blockDim.x*3] = c_v23;
	
	C += B_cols * BW;
  C[0] = c_v30;
	C[blockDim.x] = c_v31;
  C[blockDim.x*2] = c_v32;
	C[blockDim.x*3] = c_v33;
}

void matrix_multiply_unroll2(float *A, float *B, float *C, int A_rows, int A_cols, int B_cols) {
  dim3 grid(B_cols / 64, A_rows / 64);
  dim3 block(16, 16);
  k_matMult_unroll2<<< grid, block >>>(A, B, C, A_rows, A_cols, B_cols);
}

///////////////////////////////////////////////////////////////////////////////////////////////////

//
//	SGEMM routines
//
__device__ void saxpy( float a, float *b, float *c )
{
	c[0] += a*b[0];
	c[1] += a*b[1];
	c[2] += a*b[2];
	c[3] += a*b[3];
	c[4] += a*b[4];
	c[5] += a*b[5];
	c[6] += a*b[6];
	c[7] += a*b[7];
	c[8] += a*b[8];
	c[9] += a*b[9];
	c[10] += a*b[10];
	c[11] += a*b[11];
	c[12] += a*b[12];
	c[13] += a*b[13];
	c[14] += a*b[14];
	c[15] += a*b[15];
}
__global__ void sgemmNN( const float *A, int lda, const float *B,
	int ldb, float* C, int ldc, int k, float alpha, float beta )
{
	const int inx = threadIdx.x;
	const int iny = threadIdx.y;
	const int ibx = blockIdx.x * 64;
	const int iby = blockIdx.y * 16;
	const int id = inx + iny*16;
	
	A += ibx + id;
	B += inx + __mul24( iby + iny, ldb );
	C += ibx + id  + __mul24( iby, ldc );
	
	const float *Blast = B + k;
	
	float c[16] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
    
	do
	{
		float a[4] = { A[0*lda], A[1*lda], A[2*lda], A[3*lda] };

		__shared__ float bs[16][17];
		bs[inx][iny]    = B[0*ldb];
		bs[inx][iny+4]  = B[4*ldb];
		bs[inx][iny+8]  = B[8*ldb];
		bs[inx][iny+12] = B[12*ldb];
		__syncthreads();

		A += 4*lda;
		saxpy( a[0], &bs[0][0], c );		a[0] = A[0*lda];
		saxpy( a[1], &bs[1][0], c );		a[1] = A[1*lda];
		saxpy( a[2], &bs[2][0], c );		a[2] = A[2*lda];
		saxpy( a[3], &bs[3][0], c );		a[3] = A[3*lda];	

		A += 4*lda;
		saxpy( a[0], &bs[4][0], c );		a[0] = A[0*lda];
		saxpy( a[1], &bs[5][0], c );		a[1] = A[1*lda];
		saxpy( a[2], &bs[6][0], c );		a[2] = A[2*lda];
		saxpy( a[3], &bs[7][0], c );		a[3] = A[3*lda];
		
		A += 4*lda;
		saxpy( a[0], &bs[8][0], c );		a[0] = A[0*lda];
		saxpy( a[1], &bs[9][0], c );		a[1] = A[1*lda];
		saxpy( a[2], &bs[10][0], c );		a[2] = A[2*lda];
		saxpy( a[3], &bs[11][0], c );		a[3] = A[3*lda];
		
		A += 4*lda;
		saxpy( a[0], &bs[12][0], c );
		saxpy( a[1], &bs[13][0], c );
		saxpy( a[2], &bs[14][0], c );
		saxpy( a[3], &bs[15][0], c );
		
		B += 16;
		__syncthreads();
	} while( B < Blast );
	
	for( int i = 0; i < 16; i++, C += ldc )
		C[0] = alpha*c[i] + beta*C[0]; 
}	

void matrix_multiply_volvok(float *A, float *B, float *C, int A_rows, int A_cols, int B_cols) {
	dim3 grid(CEIL_DIV(A_rows, 64), CEIL_DIV(B_cols, 16));
	dim3 block(16, 4);
	
	sgemmNN<<< grid, block >>>(A, A_rows, B,
		A_cols, C, A_rows, A_cols, 1, 0 );
	//k_matMult_volvok<<< grid, block >>>(A, B, C, A_rows, A_cols, B_cols);
}

///////////////////////////////////////////////////////////////////////////////////////////////////

extern "C" void matrix_multiply_GPU(float *A, float *B, float *C, int A_rows, int A_cols, int B_cols) {
	//matrix_multiply_unroll2(A, B, C, A_rows, A_cols, B_cols);
	matrix_multiply_volvok(A, B, C, A_rows, A_cols, B_cols);
	//matrix_multiply_naive(A, B, C, A_rows, A_cols, B_cols);
}

///////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void k_fill_matrix(float *A, int M, int N, float s) {
	int row = blockDim.y * blockIdx.y + threadIdx.y;
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	if (row < M && col < N) {
		A[row * N + col] = s;
	}
}

extern "C" void fill_matrix(float *A, int M, int N, float s) {
	dim3 grid(CEIL_DIV(N, 16), CEIL_DIV(M, 16) );
	dim3 block(16, 16);
	k_fill_matrix<<< grid, block >>>(A, M, N, s);
}



