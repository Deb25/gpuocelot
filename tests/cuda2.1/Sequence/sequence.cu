/*!
	Defines a simple kernel
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#define LENGTH 11

__global__ void k_sequence(int *A, int N) {
	int n = blockDim.x * blockIdx.x + threadIdx.x;
	if (n < N) {
		A[n] = 2*n+1;
	}
}

__global__ void k_n_sequence(int *A, int N, int K) {
	int n = blockDim.x * blockIdx.x + threadIdx.x;
	for( int i = 0; i < K; ++i ) {
		if (n < N) { 
			A[n*K + i] = 2*n*K+1+i;
		}
	}
}

__global__ void k_simple_sequence(int *A) {
	int n = blockDim.x * blockIdx.x + threadIdx.x;
	A[n] = 2*n+1;
}

void simpleSequence( int* A )
{

	int* deviceA;
	
	hipMalloc( (void**) &deviceA, LENGTH * sizeof( int ) );
	hipMemcpy( deviceA, A, sizeof( int ) * LENGTH, hipMemcpyHostToDevice );
	
	k_simple_sequence<<< 1, LENGTH, 0 >>>( deviceA );
	
	hipMemcpy( A, deviceA, sizeof( int ) * LENGTH, hipMemcpyDeviceToHost );
	hipFree( deviceA );

}

void kSequence( int* A, int N )
{

	int* deviceA;
	
	hipMalloc( (void**) &deviceA, N * sizeof( int ) );
	hipMemcpy( deviceA, A, sizeof( int ) * N, hipMemcpyHostToDevice );
	unsigned int ctas = ( N % 4 == 0 ) ? N / 4 : ( N / 4 ) + 1;
	
	k_sequence<<< ctas, 4, 0 >>>( deviceA, N );
	
	hipMemcpy( A, deviceA, sizeof( int ) * N, hipMemcpyDeviceToHost );
	hipFree( deviceA );

}

void knSequence( int* A, int N, int K )
{

	int* deviceA;
	
	hipMalloc( (void**) &deviceA, K * N * sizeof( int ) );
	hipMemcpy( deviceA, A, sizeof( int ) * N * K, hipMemcpyHostToDevice );
	unsigned int ctas = ( N % 4 == 0 ) ? N / 4 : ( N / 4 ) + 1;
	
	k_n_sequence<<< 1, 32, 0 >>>( deviceA, N, K );
	
	hipMemcpy( A, deviceA, sizeof( int ) * N * K, hipMemcpyDeviceToHost );
	hipFree( deviceA );

}

int main(  )
{

	int A[ LENGTH ];

	printf( "Simple Sequence: " );

	memset( A, 0, LENGTH * sizeof( int ) );

	simpleSequence( A );

	for( unsigned int i = 0; i < LENGTH; ++i )
	{
	
		printf( "%d ", A[i] );
	
	}
	
	printf( "\n" );

	
	memset( A, 0, LENGTH * sizeof( int ) );
	
	printf( "K Sequence     : " );
	kSequence( A, LENGTH );

	for( unsigned int i = 0; i < LENGTH; ++i )
	{
	
		printf( "%d ", A[i] );
	
	}
	
	printf( "\n" );

	int B[ LENGTH * LENGTH ];
	
	memset( B, 0, LENGTH * LENGTH * sizeof( int ) );

	printf( "KN Sequence     : " );
	knSequence( B, LENGTH, LENGTH );

	for( unsigned int i = 0; i < LENGTH * LENGTH; ++i )
	{
	
		printf( "%d ", B[i] );
	
	}
	
	printf( "\nPass/Fail : Pass\n" );

}

