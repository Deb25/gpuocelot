#include "hip/hip_runtime.h"
/*!
	\file simplerFFT.cu

	\author Andrew Kerr <arkerr@gatech.edu>

	\brief implements a testbed for CUFFT
*/

//////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <hipfft/hipfft.h>
#include <cutil_inline.h>

typedef float2 Complex; 

//////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////

/*!
	Computes the discrete Fourier transform of a vector out of place
*/
void forward_DFT(Complex *input, Complex *output, int N) {
	double Pi = 3.1415926535898;
	double inv_N = 1.0 / (double)N;

	for (int k = 0; k < N; k++) {
		Complex sum = {0, 0};
		for (int n = 0; n < N; n++) {
			float w = (float)(-2.0 * Pi * inv_N * (double)k * (double)n);
			Complex x_n = input[n];
			Complex e = { cos(w), sin(w) };

			sum.x += x_n.x * e.x - x_n.y * e.y;
			sum.y += x_n.x * e.y + x_n.y * e.x;
		}
		output[k] = sum;
	}
}

/*!
	Computes the inverse discrete Fourier transform of a vector out of place
*/
void inverse_DFT(Complex *input, Complex *output, int N) {
	double Pi = 3.1415926535898;
	double inv_N = 1.0 / (double)N;

	for (int k = 0; k < N; k++) {
		Complex sum = {0, 0};
		for (int n = 0; n < N; n++) {
			float w = (float)(2.0 * Pi * inv_N * (double)k * (double)n);
			Complex x_n = input[n];
			Complex e = { cos(w), sin(w) };

			sum.x += x_n.x * e.x - x_n.y * e.y;
			sum.y += x_n.x * e.y + x_n.y * e.x;
		}
		output[k] = sum;
	}
}

/*!
	Computes the norm of two vectors. If the vector B does not exist, the norm of A is returned.
*/
double norm_difference(Complex *A, Complex *B, int N) {
	double norm_squared = 0;

	for (int n = 0; n < N; n++) {
		Complex a = A[n], b = {0, 0};
		if (B) {
			b = B[n];
		}
		a.x -= b.x;
		a.y -= b.y;
		norm_squared += a.x * a.x + a.y * a.y;
	}

	return sqrt(norm_squared);
}

//////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////

/*!
	Writes a complex vector to file data/simplefft/%s.m where %s is the vector's
	name. Defines two variables within the file, %s_len and %s. %s_len is the number
	of elements in the vector. %s is the vector itself.

	Padding with zeros is easily done without resizing the vector in memory.

	\param name name of vector and file
	\param data input data
	\param N number of elements in data
	\param padded_length length of vector as written

	\return 0 on success, -1 on I/O failure
*/
int write_vector(const char *name, Complex *data, int N, int padded_length) {
	char fname[256] = {0};
	FILE *file = 0;
	int n;

	sprintf(fname, "data/simplefft/%s.m", name);
	if (!(file = fopen(fname, "w"))) {
		fprintf(stderr, "failed to open file %s for writing\n", fname);
		return -1;
	}

	fprintf(file, "%s_len = %d;\n", name, padded_length);
	fprintf(file, "%s = [\n", name);

	for (n = 0; n < padded_length; n++) {
		if (n < N) {
			float r = data[n].x, i = data[n].y;
			fprintf(file, " %f %s %fi ;\n", r, (i >= 0 ? "+" : "-"), (float)fabs(i));
		}
		else {
			fprintf(file, " 0 ;\n");
		}
	}
	
	fprintf(file, "];\n");

	fclose(file);
	return 0;
}

/*!
	Returns a power of two not less than n.
*/
int ceiling_power_of_two(int n) {

	n |= (n >> (1<<0));
	n |= (n >> (1<<1));
	n |= (n >> (1<<2));
	n |= (n >> (1<<3));
	n |= (n >> (1<<4));

	return n + 1;
}

//////////////////////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
//////////////////////////////////////////////////////////////////////////////////////////////////
bool runTest(const int SIGNAL_LENGTH) 
{
	bool result = false;
	const size_t dataSize = sizeof(Complex ) * SIGNAL_LENGTH;

	Complex *DFT_signal_host = (Complex *)malloc(dataSize);
	Complex *T_signal_host = (Complex *)malloc(dataSize);
	Complex *T_signal_gpu = 0;
	Complex *F_signal_host = (Complex *)malloc(dataSize);
	Complex *F_signal_gpu = 0;

	hipSetDevice( cutGetMaxGflopsDeviceId() );

	hipMalloc((void **)&T_signal_gpu, dataSize);
	hipMalloc((void **)&F_signal_gpu, dataSize);

	for (int n = 0; n < SIGNAL_LENGTH; n++) {
		T_signal_host[n].x = (float)((n % 16) - 8) / (float)(SIGNAL_LENGTH - 1);
		T_signal_host[n].y = 0;
		F_signal_host[n].x = 0;
		F_signal_host[n].y = 0;
	}

	hipMemcpy(T_signal_gpu, T_signal_host, dataSize, hipMemcpyHostToDevice);
	hipMemcpy(F_signal_gpu, F_signal_host, dataSize, hipMemcpyHostToDevice);

	// compute the DFT
	forward_DFT(T_signal_host, DFT_signal_host, SIGNAL_LENGTH);

	// CUFFT plan
	hipfftHandle plan;
	cufftSafeCall(hipfftPlan1d(&plan, SIGNAL_LENGTH, HIPFFT_C2C, 1));

	// Transform signal and kernel
	cufftSafeCall(hipfftExecC2C(plan, (hipfftComplex *)T_signal_gpu, 
		(hipfftComplex *)F_signal_gpu, HIPFFT_FORWARD));

	hipMemcpy(F_signal_host, F_signal_gpu, dataSize, hipMemcpyDeviceToHost);

	double L2_error = norm_difference(DFT_signal_host, F_signal_host, SIGNAL_LENGTH);

	if (L2_error > 0.01) {
		printf("\nFFT FAILED with error %f\n", L2_error);

		write_vector("correct_F_signal", DFT_signal_host, SIGNAL_LENGTH, SIGNAL_LENGTH);
		write_vector("simple_T_signal", T_signal_host, SIGNAL_LENGTH, SIGNAL_LENGTH);
		write_vector("simple_F_signal", F_signal_host, SIGNAL_LENGTH, SIGNAL_LENGTH);
	}
	else {
		result = true;
	}

	hipFree(F_signal_gpu);
	hipFree(T_signal_gpu);

	free(T_signal_host);
	free(F_signal_host);
	free(DFT_signal_host);

	return result;
}

//////////////////////////////////////////////////////////////////////////////////////////////////
// Program main
//////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char** argv) {
	bool result = true;

	int size = 0;

	if (argc > 1) {
		size = atoi(argv[1]);
	}

	if (size) {
		result = runTest(size);
		if (result) {
			printf("TEST PASSED\nFFT size %d passed\n", size);
		}
		else {
			printf("TEST FAILED on FFT size %d\n", size);
		}
	}
	else {
		result = runTest(1024);

		if (result) {
			printf("TEST PASSED\n2^10 FFT size passed\n");
		}
	}

	return 0;
}

//////////////////////////////////////////////////////////////////////////////////////////////////

